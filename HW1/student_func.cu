#include "hip/hip_runtime.h"
#include "student_func.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  int row = threadIdx.x;
  int col = blockIdx.x;
  int index = numCols * row + col;

  greyImage[index] = 0.299f * (float)rgbaImage[index].x + 0.587f * (float)rgbaImage[index].y + 0.114f * (float)rgbaImage[index].z;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  const dim3 blockSize(numRows, 1, 1);  //TODO
  const dim3 gridSize(numCols, 1, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
  